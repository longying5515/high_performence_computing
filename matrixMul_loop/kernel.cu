﻿
#include <hip/hip_runtime.h>
#include <iostream>
#include <hipblas.h>

void matrixMultiplication(const hipblasHandle_t& handle, int size) {
    const int n = size;

    // 分配主机内存
    float* h_A = new float[n * n];
    float* h_B = new float[n * n];
    float* h_C = new float[n * n];

    // 初始化矩阵
    // TODO: 根据实际需求初始化矩阵 h_A 和 h_B

    // 分配设备内存
    float* d_A, * d_B, * d_C;
    hipMalloc((void**)&d_A, n * n * sizeof(float));
    hipMalloc((void**)&d_B, n * n * sizeof(float));
    hipMalloc((void**)&d_C, n * n * sizeof(float));

    // 将数据从主机内存复制到设备内存
    hipMemcpy(d_A, h_A, n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, n * n * sizeof(float), hipMemcpyHostToDevice);

    // 执行矩阵乘法
    const float alpha = 1.0f;
    const float beta = 0.0f;
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, d_A, n, d_B, n, &beta, d_C, n);

    // 将结果从设备内存复制回主机内存
    hipMemcpy(h_C, d_C, n * n * sizeof(float), hipMemcpyDeviceToHost);

    // TODO: 处理矩阵相乘结果

    // 释放内存
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

int main() {
    // 初始化 CUBLAS
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // 矩阵规模从 512 增加至 8192（可以根据需要调整）
    for (int size = 512; size <= 8192; size *= 2) {
        std::cout << "Matrix Size: " << size << "x" << size << std::endl;
        matrixMultiplication(handle, size);
        std::cout << std::endl;
    }

    // 销毁 CUBLAS 句柄
    hipblasDestroy(handle);

    return 0;
}
