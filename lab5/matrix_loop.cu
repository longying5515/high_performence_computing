
#include <hip/hip_runtime.h>
#include <iostream>
#include <hipblas.h>
#include <random>
using namespace std;
void matrixMultiplication(const hipblasHandle_t& handle, int size) {
    const int n = size;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // 分配主机内存
    float *h_A = new float[n * n];
    float *h_B = new float[n * n];
    float *h_C = new float[n * n];

    // 初始化矩阵
    random_device rd;
    mt19937 gen(rd());
    uniform_real_distribution<float> dis(0.0f, 1.0f);

    for (int i = 0; i < n * n; ++i) {
        h_A[i] = dis(gen);
        h_B[i] = dis(gen);
    }

    // 分配设备内存
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, n * n * sizeof(float));
    hipMalloc((void**)&d_B, n * n * sizeof(float));
    hipMalloc((void**)&d_C, n * n * sizeof(float));

    // 将数据从主机内存复制到设备内存
    hipMemcpy(d_A, h_A, n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, n * n * sizeof(float), hipMemcpyHostToDevice);

    // 执行矩阵乘法
    const float alpha = 1.0f;
    const float beta = 0.0f;
    hipEventRecord(start);
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, d_A, n, d_B, n, &beta, d_C, n);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    // 将结果从设备内存复制回主机内存
    hipMemcpy(h_C, d_C, n * n * sizeof(float), hipMemcpyDeviceToHost);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);        
    cout << "Matrix Size:"<<size<<"  Time:"<<milliseconds<<"ms"<<endl;
    // TODO: 处理矩阵相乘结果

    // 释放内存
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

int main() {
    // 初始化 CUBLAS
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    // 矩阵规模从 512 增加至 8192（可以根据需要调整）
    for (int size = 512; size <= 8192; size *= 2) {
        matrixMultiplication(handle, size);
    }

    // 销毁 CUBLAS 句柄
    hipblasDestroy(handle);

    return 0;
}